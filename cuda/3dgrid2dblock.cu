/* File: 3dgrid2dblock.cu
 * Name: D.Saravanan
 * Date: 25/11/2024
 * Program compute thread index with 3D grid of 2D blocks
 *
 * $ nvcc -o 3dgrid2dblock 3dgrid2dblock.cu
 * $ ./3dgrid2dblock
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

/* 3D grid of 2D blocks */
__global__ void threadId_3D_2D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x \
                  + gridDim.x * gridDim.y * blockIdx.z;

    int threadId = blockId * (blockDim.x * blockDim.y) \
                   + (threadIdx.y * blockDim.x) + threadIdx.x;

    printf("blockId: %d = blockIdx.x: %d + blockIdx.y: %d * gridDim.x: %d "
           "+ gridDim.x: %d * gridDim.y: %d * blockIdx.x: %d\n", blockId,\
           blockIdx.x, blockIdx.y, gridDim.x, gridDim.x, gridDim.y, blockIdx.z);
    printf("\n");
    printf("threadId: %d = blockId: %d + (blockDim.x: %d * blockDim.y: %d) "
           "+ (threadIdx.y: %d * blockDim.x: %d) + threadIdx.x: %d\n", threadId,\
           blockId, blockDim.x, blockDim.y, threadIdx.y, blockDim.x, threadIdx.x);
}

int main() {
    hipSetDevice(0);

    dim3 gridDim, blockDim;

    gridDim.x = 2;
    gridDim.y = 3;
    gridDim.z = 2;

    blockDim.x = 4;
    blockDim.y = 2;

    threadId_3D_2D<<<gridDim, blockDim>>>();

    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}
