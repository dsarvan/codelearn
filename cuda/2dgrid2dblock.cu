/* File: 2dgrid2dblock.cu
 * Name: D.Saravanan
 * Date: 22/11/2024
 * Program compute thread index with 2D grid of 2D blocks
 *
 * $ nvcc -o 2dgrid2dblock 2dgrid2dblock.cu
 * $ ./2dgrid2dblock
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

/* 2D grid of 2D blocks */
__global__ void threadId_2D_2D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) \
                   + (threadIdx.y * blockDim.x) + threadIdx.x;

    printf("blockId: %d = blockIdx.x: %d + blockIdx.y: %d * gridDim.x: %d\n",\
            blockId, blockIdx.x, blockIdx.y, gridDim.x);
    printf("\n");
    printf("threadId: %d = blockId: %d * (blockDim.x: %d * blockDim.y: %d) "
           "+ (threadIdx.y: %d * blockDim.x: %d) + threadIdx.x: %d\n", threadId,\
           blockId, blockDim.x, blockDim.y, threadIdx.y, blockDim.x, threadIdx.x);
}

int main() {
    hipSetDevice(0);

    dim3 gridDim, blockDim;

    gridDim.x = 2;
    gridDim.y = 3;

    blockDim.x = 4;
    blockDim.y = 2;

    threadId_2D_2D<<<gridDim, blockDim>>>();

    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}
