/* File: 3dgrid1dblock.cu
 * Name: D.Saravanan
 * Date: 24/11/2024
 * Program compute thread index with 3D grid of 1D blocks
 *
 * $ nvcc -o 3dgrid1dblock 3dgrid1dblock.cu
 * $ ./3dgrid1dblock
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

/* 3D grid of 1D blocks */
__global__ void threadId_3D_1D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x \
                  + gridDim.x * gridDim.y * blockIdx.z;

    int threadId = blockId * blockDim.x + threadIdx.x;

    printf("blockId: %d = blockIdx.x: %d + blockIdx.y: %d * gridDim.x: %d "
           "+ gridDim.x: %d * gridDim.y: %d * blockIdx.z: %d\n", blockId,\
           blockIdx.x, blockIdx.y, gridDim.x, gridDim.x, gridDim.y, blockIdx.z);
    printf("\n");
    printf("threadId: %d = blockId: %d * blockDim.x: %d + threadIdx.x: %d\n",\
            threadId, blockId, blockDim.x, threadIdx.x);
}

int main() {
    hipSetDevice(0);

    dim3 gridDim, blockDim;

    gridDim.x = 2;
    gridDim.y = 3;
    gridDim.z = 2;

    blockDim.x = 4;

    threadId_3D_1D<<<gridDim, blockDim>>>();

    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}
